// Host Side Code for Cross-correlation in GPU

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <fstream>
//#include "Cross_Data_type.h"
#include "corr2Mex.h"
#include "normXcorr_GPUKernel.cu"

using namespace std;


Matrix AllocateDeviceMatrix(const Matrix M);
Matrix AllocateMatrix(int height, int width,int init);
void CopyToDeviceMatrix(Matrix Mdevice, const Matrix Mhost);
void CopyFromDeviceMatrix(Matrix Mhost, const Matrix Mdevice);
bool CompareResults(float* A, float* B, int elements, float eps);
void FreeDeviceMatrix(Matrix* M);
void FreeMatrix(Matrix* M);

void CorrelationOnDevice(const Matrix M, const Matrix N, float *CorrH, params parameters);


int main(int argc,char** argv) {
	// Input Parameters
	
	if(argc!=9)
	{
		printf("Usage %s Parameters missing\n",argv[0]);
		return 1;
	}
	
	int imageWidth = atoi(argv[1]);
	int imageHeight = atoi(argv[2]);
	int SEARCH_X = atoi(argv[3]);
	int SEARCH_Y = atoi(argv[4]);
	int KERNEL_X = atoi(argv[5]);
	int KERNEL_Y = atoi(argv[6]);
	int numX = atoi(argv[7]);
	int numY = atoi(argv[8]);
	int DisplacementSize = numX*numY;
	int Corr_size = SEARCH_X*SEARCH_Y;
	Matrix Pre;
	Matrix Post;
	
	float OVERLAP = 50.0;
	
	params parameters = {SEARCH_Y,SEARCH_X,KERNEL_Y,KERNEL_X,OVERLAP,numX,numY};
	Pre  = AllocateMatrix(imageHeight,imageWidth, 1);
	Post  = AllocateMatrix(imageHeight,imageWidth, 1);	
	float gpuTime=0.f;
	
	// Allocating Host-side Memory for Cross-correlation
	/*SoA_Corr *CorrH;
	CorrH = (SoA_Corr *)malloc(sizeof(SoA_Corr)*DisplacementSize);
	for(int k=0;k<DisplacementSize;k++){
		CorrH[k].Corr_Points = (float*) malloc(Corr_size*sizeof(float));
	}*/
	float *CorrH;
	CorrH = (float*)malloc(Corr_size*DisplacementSize*sizeof(float));
	

	float  elapsedTime_inc;
	hipEvent_t startEvent_inc, stopEvent_inc;
	hipEventCreate(&startEvent_inc);
	hipEventCreate(&stopEvent_inc);
	hipEventRecord(startEvent_inc,0); // starting timing for inclusive  
	
	CorrelationOnDevice(Pre, Post, CorrH, parameters); // Execution Model for GPU is set up in this function

	
    hipEventRecord(stopEvent_inc,0);  //ending timing for inclusive
	hipEventSynchronize(stopEvent_inc);   
	hipEventElapsedTime(&elapsedTime_inc, startEvent_inc, stopEvent_inc);
	gpuTime = elapsedTime_inc;
	
	// Printing Cross-correlation Matrix for Block:0
	for(int h=0;h<DisplacementSize;h++){
		for(int z=0;z<SEARCH_X;z++){
			for(int g=0;g<SEARCH_Y;g++){
				printf("%f ",CorrH[g+SEARCH_X*(z+SEARCH_Y*h)]);
			}
		printf("\n");	
		}
		printf("\n");
	}
	
	printf("\n");
	
	// Free matrices
	FreeMatrix(&Pre);
	FreeMatrix(&Post);
	return 0;
	
}

//// Cuda Kernel Call //////

void CorrelationOnDevice(const Matrix Pre, const Matrix Post, float *CorrH, params parameters)
{
	// Load Pre and Post to the device
	Matrix Pred = AllocateDeviceMatrix(Pre);
	CopyToDeviceMatrix(Pred, Pre);
	Matrix Postd = AllocateDeviceMatrix(Post);
	CopyToDeviceMatrix(Postd, Post);
	
    
	// Allocate Space for Pre-Mean
	float *preMean;
	float *preVar;
	hipMalloc((void **)&preMean,sizeof(float)*parameters.numX*parameters.numY);
	hipMalloc((void **)&preVar,sizeof(float)*parameters.numX*parameters.numY);
	
	// Allocate SoA on the device ?????
	float *CorrD;
	hipMalloc((void **)&CorrD,sizeof(float)*parameters.numX*parameters.numY*parameters.searchX*parameters.searchY);
	
	//hipMalloc((SoA_Corr **)&CorrD,sizeof(SoA_Corr)*parameters.numX*parameters.numY);

	// Setup the execution configuration

	dim3 dimBlock(parameters.searchX, parameters.searchY);
	//dim3 dimBlock(2*parameters.searchX, 2*parameters.searchY);
	dim3 dimGrid(parameters.numX, parameters.numY); 
	int sharedmemsize = 2*parameters.searchX*parameters.searchY*sizeof(float);
	// Launch the device computation threads!
	
    normXcorr_GPU<<<dimGrid, dimBlock,sharedmemsize>>>(Pred,Postd,CorrD,parameters,preMean,preVar);
  
	//Copting SoA from Device to Host
	//CopyFromDeviceMatrix(Corr, Corrd); 
	//hipMemcpy(CorrH,CorrD,sizeof(SoA_Corr)*parameters.numX*parameters.numY,hipMemcpyDeviceToHost);
	hipMemcpy(CorrH,CorrD,sizeof(float)*parameters.numX*parameters.numY*parameters.searchX*parameters.searchY,hipMemcpyDeviceToHost);
	
	// Free device matrices
	FreeDeviceMatrix(&Pred);
	FreeDeviceMatrix(&Postd);
	hipFree(CorrD);
	//FreeDeviceMatrix(&Corrd);

}



// Allocate a device matrix of same size as M.
Matrix AllocateDeviceMatrix(const Matrix M)
{
	Matrix Mdevice = M;
	int size = M.width * M.height * sizeof(float);
	hipMalloc((void**)&Mdevice.elements, size);
	return Mdevice;
}

Matrix AllocateMatrix(int height, int width,int init) // 1 is file read/ 0 is just allocation
{
	Matrix M;
	M.width = M.pitch = width;
	M.height = height;
	int size = M.width * M.height;
	M.elements = NULL;
	FILE *fp;
	fp = fopen("trialNumbers.inp","r");
	// don't allocate memory on option 2
	M.elements = (float*) malloc(size*sizeof(float));
	
	if(init)
	{
		for(unsigned int i = 0; i < M.width * M.height; i++)
		{
			fscanf(fp,"%f",&M.elements[i]);
		}
	}
	return M;
}	

// Copy a host matrix to a device matrix.
void CopyToDeviceMatrix(Matrix Mdevice, const Matrix Mhost)
{
	int size = Mhost.width * Mhost.height * sizeof(float);
	Mdevice.height = Mhost.height;
	Mdevice.width = Mhost.width;
	Mdevice.pitch = Mhost.pitch;
	hipMemcpy(Mdevice.elements, Mhost.elements, size, 
			hipMemcpyHostToDevice);
}

// Copy a device matrix to a host matrix.
void CopyFromDeviceMatrix(Matrix Mhost, const Matrix Mdevice)
{
	int size = Mdevice.width * Mdevice.height * sizeof(float);
	hipMemcpy(Mhost.elements, Mdevice.elements, size, 
			hipMemcpyDeviceToHost);
}

// Free a device matrix.
void FreeDeviceMatrix(Matrix* M)
{
	hipFree(M->elements);
	M->elements = NULL;
}

// Free a host Matrix
void FreeMatrix(Matrix* M)
{
	free(M->elements);
	M->elements = NULL;
}



